
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

//sample function, mostly useless
extern "C" 
__global__ void saxpy(__half a, __half *x, __half *y, __half *out, size_t n){
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        out[tid] = a * x[tid] + y[tid];
    }
}

extern "C" 
__global__ void pairwise(size_t dim_row, size_t dim_col, size_t dim_window,
                         unsigned short int *x, unsigned short int *y, unsigned short int *out)
{
    unsigned int foci = x[blockIdx.x];
    size_t num_row = (size_t)(blockIdx.x / dim_row);
    size_t row_idx = blockIdx.x % dim_row;
    size_t locationY = row_idx + num_row * (dim_row + dim_window);
    size_t offset = (dim_row + dim_window) * (size_t)(threadIdx.x / dim_row)
                    + (threadIdx.x % dim_row);
    unsigned int target = y[locationY + offset];
    unsigned short int residual = (__vabsdiffu2(foci, target));
    out[blockIdx.x * blockDim.x + threadIdx.x] = residual;
}

extern "C" 
__global__ void pairwise_line(size_t dim_row, size_t dim_col, size_t dim_window,
                         unsigned short int *x, unsigned short int *y, unsigned short int *out)
{
    unsigned int bank = threadIdx.x / dim_window;
    unsigned int foci_idx = (12 * blockIdx.x) + bank;
    unsigned int foci = x[foci_idx];

    unsigned int row_idx = ((threadIdx.x % dim_window) + (foci_idx / dim_row)) * (dim_row + dim_window);
    unsigned int row_offset = foci_idx % dim_row;
    unsigned int coalesce_idx = threadIdx.x + (foci_idx / dim_row) * (dim_row + dim_window);
    
    for (int i = 0; i < dim_window; i ++){
        //unsigned int target = y[coalesce_idx + (i * dim_window)];
        unsigned int target = y[row_idx + row_offset + i];
        unsigned short int residual = (__vabsdiffu2(foci, target));
        out[blockIdx.x * blockDim.x + threadIdx.x + i] = residual;
    }
}

extern "C" 
__global__ void pairwise_line_alt(size_t dim_row, size_t dim_col, size_t dim_window,
                         unsigned short int *x, unsigned short int *y, unsigned short int *out)
{

}