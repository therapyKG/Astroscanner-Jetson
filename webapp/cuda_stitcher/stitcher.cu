//#include "stitcher.h"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

extern "C" 
__global__ void saxpy(__half a, __half *x, __half *y, __half *out, size_t n){
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        out[tid] = a * x[tid] + y[tid];
    }
}

extern "C" 
__global__ void pairwise(size_t dim_row, size_t dim_col, size_t dim_window,
                         unsigned short int *x, unsigned short int *y, unsigned short int *out)
{
    unsigned int foci = (unsigned int)x[blockIdx.x];
    size_t num_row = (size_t)(blockIdx.x / dim_row);
    size_t row_idx = blockIdx.x % dim_row;
    size_t locationY = row_idx + num_row * (dim_row + dim_window);
    size_t offset = (dim_row + dim_window) * (size_t)(threadIdx.x / dim_row)
                    + (threadIdx.x % dim_row);
    unsigned int target = (unsigned int)y[locationY + offset];
    unsigned short int residual = (unsigned short int)(__vabsdiffu2(foci, target));
    out[blockIdx.x * blockDim.x + threadIdx.x] = residual;
}